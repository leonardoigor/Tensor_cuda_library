#include "hip/hip_runtime.h"
#include "include/tensor.cuh"
#include "tensor.cuh"
// #include "tensor.cuh"
// #include "tensor.cuh"
// #include "tensor.cuh"

Tensor::Tensor(int rows, int cols) : rows(rows), cols(cols)
{
    hipMalloc(&data, rows * cols * sizeof(float));
}

Tensor::~Tensor()
{

    hipFree(data);
}
// Copy constructor
Tensor::Tensor(const Tensor &other) : rows(other.rows), cols(other.cols)
{
    hipMalloc(&data, rows * cols * sizeof(float));
    hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
}

__host__ Tensor &Tensor::operator=(const Tensor &other)
{
    if (this != &other)
    {
        hipFree(data);
        rows = other.rows;
        cols = other.cols;
        hipMalloc(&data, rows * cols * sizeof(float));
        hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
    }
    return *this;
}
__host__ int Tensor::getRows() const
{
    return rows;
}
__host__ int Tensor::getCols() const
{
    return cols;
}

__host__ void Tensor::setValues(float *hostData)
{
    hipMemcpy(data, hostData, rows * cols * sizeof(float), hipMemcpyHostToDevice);
}

__host__ void Tensor::print() const
{
    float *hostData = new float[rows * cols];
    hipMemcpy(hostData, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            std::cout << hostData[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    delete[] hostData;
}

__host__ void Tensor::toCPU(float *hostData) const
{
    hipMemcpy(hostData, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
}

__device__ float &Tensor::operator()(int i, int j) const
{
    return data[i * cols + j];
}

__global__ void MULGLOAL(float *a, float *b, float *c, int cols, int rows)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; ++j)
        {
            int tid = blockIdx.x * blockDim.x + threadIdx.x;
            if (tid < rows * cols)
            {
                c[tid] = a[tid] * b[tid];
                // printf("\n----%d,%d--(%f)-\n", j, i, c[tid]);
            }
        }
    }
    // printf("\n----%d,%d---\n", cols, rows);
}

__host__ void Tensor::Mul(Tensor *b, Tensor *c)
{
    printf("Test\n");
    auto cols = this->cols;
    auto rows = this->rows;
    int blockSize = 256;
    int numBlocks = (rows * cols + blockSize - 1) / blockSize;
    // c->print();

    MULGLOAL<<<numBlocks, blockSize>>>(this->data, b->data, c->data, cols, rows);
    hipDeviceSynchronize();
    // c->print();
    // hipError_t hipError_t = hipGetLastError();
    // if (hipError_t != hipSuccess)
    // {
    //     std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
    // }
    // else
    // {
    //     std::cout << "No CUDA error detected." << std::endl;
    // }
}
// __device__ Tensor Tensor::operator+(const Tensor &other) const
// {
//     Tensor result(rows, cols);
//     for (int i = 0; i < rows; ++i)
//     {
//         for (int j = 0; j < cols; ++j)
//         {
//             result(i, j) = (*this)(i, j) + other(i, j);
//         }
//     }
//     return result;
// }

// __device__ Tensor Tensor::operator-(const Tensor &other) const
// {
//     Tensor result(*this);
//     for (int i = 0; i < rows; ++i)
//     {
//         for (int j = 0; j < cols; ++j)
//         {
//             result(i, j) = (*this)(i, j) - other(i, j);
//         }
//     }
//     return result;
// }

Tensor Tensor::operator*(Tensor &other)
{
    Tensor result(rows, other.cols);
    Tensor *b = &other;
    Tensor *result2 = &result;
    Mul(b, result2);
    return result;
}
