#include "hip/hip_runtime.h"
#include "include/Tensor.cuh"
#include "Tensor.cuh"
// #include "Tensor1d.cuh"
// #include "Tensor1d.cuh"
// #include "Tensor1d.cuh"

Tensor1d::Tensor1d(int rows, int cols) : rows(rows), cols(cols)
{
    hipMalloc(&data, rows * cols * sizeof(float));
}

Tensor1d::~Tensor1d()
{

    hipFree(data);
}
// Copy constructor
Tensor1d::Tensor1d(const Tensor1d &other) : rows(other.rows), cols(other.cols)
{
    hipMalloc(&data, rows * cols * sizeof(float));
    hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
}

__host__ Tensor1d &Tensor1d::operator=(const Tensor1d &other)
{
    if (this != &other)
    {
        hipFree(data);
        rows = other.rows;
        cols = other.cols;
        hipMalloc(&data, rows * cols * sizeof(float));
        hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
    }
    return *this;
}
__host__ int Tensor1d::getRows() const
{
    return rows;
}
__host__ int Tensor1d::getCols() const
{
    return cols;
}

__host__ void Tensor1d::setValues(float *hostData)
{
    hipMemcpy(data, hostData, rows * cols * sizeof(float), hipMemcpyHostToDevice);
}

__host__ void Tensor1d::print() const
{
    float *hostData = new float[rows * cols];
    hipMemcpy(hostData, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            std::cout << hostData[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    delete[] hostData;
}

__host__ void Tensor1d::toCPU(float *hostData) const
{
    hipMemcpy(hostData, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
}

__device__ float &Tensor1d::operator()(int i, int j) const
{
    return data[i * cols + j];
}

__global__ void MULGLOAL(float *a, float *b, float *c, int cols, int rows)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; ++j)
        {
            int tid = blockIdx.x * blockDim.x + threadIdx.x;
            if (tid < rows * cols)
            {
                c[tid] = a[tid] * b[tid];
                // printf("\n----%d,%d--(%f)-\n", j, i, c[tid]);
            }
        }
    }
    // printf("\n----%d,%d---\n", cols, rows);
}
__global__ void SUMGLOAL(float *a, float *b, float *c, int cols, int rows)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; ++j)
        {
            int tid = blockIdx.x * blockDim.x + threadIdx.x;
            if (tid < rows * cols)
            {
                // printf("d %f\n", c[tid]);
                c[tid] = a[tid] + b[tid];
                // printf("a %f\n", c[tid]);
            }
        }
    }
}

__host__ void Tensor1d::Mul(Tensor1d *b, Tensor1d *c)
{
    // printf("Test\n");
    auto cols = this->cols;
    auto rows = this->rows;
    int blockSize = 256;
    int numBlocks = (rows * cols + blockSize - 1) / blockSize;
    // c->print();

    MULGLOAL<<<numBlocks, blockSize>>>(this->data, b->data, c->data, cols, rows);
    hipDeviceSynchronize();
}
__host__ void Tensor1d::Sum(Tensor1d *b, Tensor1d *c)
{
    // printf("Test\n");
    auto cols = this->cols;
    auto rows = this->rows;
    int blockSize = 256;
    int numBlocks = (rows * cols + blockSize - 1) / blockSize;
    // c->print();

    SUMGLOAL<<<numBlocks, blockSize>>>(this->data, b->data, c->data, cols, rows);
    hipDeviceSynchronize();
}
// c->print();
// hipError_t hipError_t = hipGetLastError();
// if (hipError_t != hipSuccess)
// {
//     std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
// }
// else
// {
//     std::cout << "No CUDA error detected." << std::endl;
// }
Tensor1d Tensor1d::operator+(Tensor1d &other)
{
    Tensor1d result(rows, cols);
    Tensor1d *b = &other;
    Tensor1d *result2 = &result;
    Sum(b, result2);
    return result;
}

// __device__ Tensor1d Tensor1d::operator-(const Tensor1d &other) const
// {
//     Tensor1d result(*this);
//     for (int i = 0; i < rows; ++i)
//     {
//         for (int j = 0; j < cols; ++j)
//         {
//             result(i, j) = (*this)(i, j) - other(i, j);
//         }
//     }
//     return result;
// }

Tensor1d Tensor1d::operator*(Tensor1d &other)
{
    Tensor1d result(rows, other.cols);
    Tensor1d *b = &other;
    Tensor1d *result2 = &result;
    Mul(b, result2);
    return result;
}
