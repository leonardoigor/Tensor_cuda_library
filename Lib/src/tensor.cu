#include "include/tensor.cuh"

Tensor::Tensor(int rows, int cols) : rows(rows), cols(cols)
{
    hipMalloc(&data, rows * cols * sizeof(float));
}

Tensor::~Tensor()
{

    std::cout << "Freeing data" << std::endl;
    hipFree(data);
}
// Copy constructor
Tensor::Tensor(const Tensor &other) : rows(other.rows), cols(other.cols)
{
    hipMalloc(&data, rows * cols * sizeof(float));
    hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
}
// Assignment operator
Tensor &Tensor::operator=(const Tensor &other)
{
    if (this != &other)
    {
        hipFree(data);
        rows = other.rows;
        cols = other.cols;
        hipMalloc(&data, rows * cols * sizeof(float));
        hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
    }
    return *this;
}
