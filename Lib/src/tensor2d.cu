#include "hip/hip_runtime.h"
#include "include/tensor2d.cuh"
#include "tensor2d.cuh"
#include "tensor2d_kernel.cuh"

Tensor2d::Tensor2d(int cols, int rows) : rows(rows), cols(cols)
{
}

Tensor2d Tensor2d::Random(int min, int max, int cols, int rows)
{
    Tensor2d result(cols, rows);
    float **h_data = (float **)malloc(rows * sizeof(float *));
    for (int i = 0; i < rows; i++)
    {
        h_data[i] = (float *)malloc(cols * sizeof(float));
    }
    float **d_data;
    hipMalloc((void **)&d_data, rows * sizeof(float *));
    for (int i = 0; i < rows; i++)
    {
        hipMalloc((void **)&d_data[i], cols * sizeof(float));
    }

    dim3 blockDim(16, 16);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y);
    RandomInRangeKernel<<<gridDim, blockDim>>>(min, max, cols, rows, d_data);
    hipDeviceSynchronize();

    // Copy the randomized data back to host memory
    hipMemcpy(h_data, d_data, rows * sizeof(float *), hipMemcpyDeviceToHost);
    for (int i = 0; i < rows; i++)
    {
        hipMemcpy(h_data[i], d_data[i], cols * sizeof(float), hipMemcpyDeviceToHost);
    }
    return result;
}

Tensor2d::~Tensor2d()
{
}