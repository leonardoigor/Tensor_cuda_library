#include "hip/hip_runtime.h"
#include "include/tensor2d.cuh"
#include "tensor2d.cuh"
#include "tensor2d_kernel.cuh"
#include "iostream"
#include <random>
#include <hiprand/hiprand_kernel.h>
#include "utils.cuh"

Tensor2d::Tensor2d(int cols, int rows) : rows(rows), cols(cols)
{
    CUDA_CHECK(hipMalloc((void **)&data, cols * rows * sizeof(double)));
}

Tensor2d::~Tensor2d()
{
    hipFree(data);
}
void Tensor2d::print()
{

    double *hostData = new double[rows * cols];
    hipMemcpy(hostData, data, rows * cols * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "[";
    for (int i = 0; i < rows; ++i)
    {
        std::cout << "[";
        for (int j = 0; j < cols; ++j)
        {
            std::cout << hostData[i * cols + j];
            if (j < cols - 1)
            {
                std::cout << ",";
            }
        }
        std::cout << "]";
        if (i < rows - 1)
        {
            std::cout << ",";
        }
    }
    std::cout << "]" << std::endl;
    delete[] hostData;
}
Tensor2d *Tensor2d::Random(int min, int max, int cols, int rows)
{
    Tensor2d *result = new Tensor2d(cols, rows);
    double *data_d = (double *)malloc(cols * rows * sizeof(double));

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;

    CreateRandomListMinMax<<<blocksPerGrid, threadsPerBlock>>>(min, max, cols, rows, result->data);

    hipDeviceSynchronize();
    return result;
}

Tensor2d *Tensor2d::mul(double a)
{
    // Define grid and block dimensions
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;
    MultiplyData<<<blocksPerGrid, threadsPerBlock>>>(a, cols, rows, data);
    hipDeviceSynchronize();
    return this;
}
Tensor2d *Tensor2d::setValue(double a)
{
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;
    SetData<<<blocksPerGrid, threadsPerBlock>>>(a, cols, rows, data);
    hipDeviceSynchronize();
    return this;
}