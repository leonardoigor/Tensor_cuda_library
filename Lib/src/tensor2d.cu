#include "hip/hip_runtime.h"
#include "include/tensor2d.cuh"
#include "tensor2d.cuh"
#include "tensor2d_kernel.cuh"
#include "iostream"
#include <random>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CHECK(call)                                                              \
    do                                                                                \
    {                                                                                 \
        hipError_t error = call;                                                     \
        if (error != hipSuccess)                                                     \
        {                                                                             \
            printf("CUDA error at %s:%d code=%d(%s) \"%s\" \n",                       \
                   __FILE__, __LINE__, (int)error, hipGetErrorString(error), #call); \
            exit(EXIT_FAILURE);                                                       \
        }                                                                             \
    } while (0)

// Function to generate random number between min and max
__device__ double generateRandomNumber(double min, double max, hiprandState_t *state)
{
    // Generate random number within [min, max] range
    return min + (max - min) * hiprand_uniform_double(state);
}
__global__ void CreateRandomListMinMax(int min, int max, int cols, int rows, double *data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState_t state;
    hiprand_init(clock64(), tid, 0, &state);
    for (int i = tid; i < rows * cols; i += stride)
    {
        auto value = generateRandomNumber(min, max, &state);
        data[i] = value;
    }
}
// Kernel to multiply each element of data with a double value
__global__ void MultiplyData(double multiplier, int cols, int rows, double *data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < rows * cols; i += stride)
    {
        data[i] *= multiplier;
    }
}
__global__ void SetData(double val, int cols, int rows, double *data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < rows * cols; i += stride)
    {
        data[i] = val;
    }
}

Tensor2d::Tensor2d(int cols, int rows) : rows(rows), cols(cols)
{
    CUDA_CHECK(hipMalloc((void **)&data, cols * rows * sizeof(double)));
}

Tensor2d::~Tensor2d()
{
    hipFree(data);
}
void Tensor2d::print()
{

    double *hostData = new double[rows * cols];
    hipMemcpy(hostData, data, rows * cols * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "[";
    for (int i = 0; i < rows; ++i)
    {
        std::cout << "[";
        for (int j = 0; j < cols; ++j)
        {
            std::cout << hostData[i * cols + j];
            if (j < cols - 1)
            {
                std::cout << ",";
            }
        }
        std::cout << "]";
        if (i < rows - 1)
        {
            std::cout << ",";
        }
    }
    std::cout << "]" << std::endl;
    delete[] hostData;
}
Tensor2d *Tensor2d::Random(int min, int max, int cols, int rows)
{
    Tensor2d *result = new Tensor2d(cols, rows);
    double *data_d = (double *)malloc(cols * rows * sizeof(double));

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;

    CreateRandomListMinMax<<<blocksPerGrid, threadsPerBlock>>>(min, max, cols, rows, result->data);

    hipDeviceSynchronize();
    return result;
}

Tensor2d *Tensor2d::mul(double a)
{
    // Define grid and block dimensions
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;
    MultiplyData<<<blocksPerGrid, threadsPerBlock>>>(a, cols, rows, data);
    hipDeviceSynchronize();
    return this;
}
Tensor2d *Tensor2d::setValue(double a)
{
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;
    SetData<<<blocksPerGrid, threadsPerBlock>>>(a, cols, rows, data);
    hipDeviceSynchronize();
    return this;
}