#include "hip/hip_runtime.h"
#include "include/tensor2d.cuh"
#include "tensor2d.cuh"
#include "tensor2d_kernel.cuh"
#include "iostream"
#include <random>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CHECK(call)                                                              \
    do                                                                                \
    {                                                                                 \
        hipError_t error = call;                                                     \
        if (error != hipSuccess)                                                     \
        {                                                                             \
            printf("CUDA error at %s:%d code=%d(%s) \"%s\" \n",                       \
                   __FILE__, __LINE__, (int)error, hipGetErrorString(error), #call); \
            exit(EXIT_FAILURE);                                                       \
        }                                                                             \
    } while (0)

// Function to generate random number between min and max
__device__ double generateRandomNumber(double min, double max, hiprandState_t *state)
{
    // Generate a random number between 0 and 1
    double randNum = hiprand_uniform(state);

    // Scale and shift the random number to fit within the desired range
    return min + randNum * (max - min);
}
__global__ void CreateRandomListMinMax(int min, int max, int cols, int rows, double *data)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = 0; i < rows * cols; i++)
    {
        hiprandState_t state;
        hiprand_init(0, stride, 0, &state);

        data[i] = generateRandomNumber(min, max, &state);
    }
}

Tensor2d::Tensor2d(int cols, int rows) : rows(rows), cols(cols)
{
}

Tensor2d Tensor2d::Random(int min, int max, int cols, int rows)
{
    Tensor2d result(cols, rows);
    double *data_d = (double *)malloc(cols * rows * sizeof(double));

    CUDA_CHECK(hipMalloc(&result.data, cols * rows * sizeof(double)));
    int threadsPerBlock = 256;
    int blocksPerGrid = ((cols * rows) + threadsPerBlock - 1) / threadsPerBlock;

    CreateRandomListMinMax<<<blocksPerGrid, threadsPerBlock>>>(min, max, cols, rows, result.data);

    return result;
}

Tensor2d::~Tensor2d()
{
    hipFree(data);
}
void Tensor2d::Print()
{
}