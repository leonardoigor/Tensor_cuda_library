#include "hip/hip_runtime.h"
#include "tensor2d_kernel.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void RandomInRangeKernel(int min, int max, int cols, int rows, float **results)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols)
    {
        hiprandState state;
        hiprand_init(0, row * cols + col, 0, &state); // Initialize hiprand state

        // // Generate a random value within the specified range
        printf("[(%d,%d) (%d,%d)] ", row, rows, col, cols);
        float random_value = min + (max - min) * hiprand_uniform(&state);
        // results[row][col] = random_value;
        printf(" r= (%f,%f)", random_value, results[col][row]);
    }
}
// Function to generate random number between min and max
__device__ double generateRandomNumber(double min, double max, hiprandState_t *state)
{
    // Generate random number within [min, max] range
    return min + (max - min) * hiprand_uniform_double(state);
}
__global__ void CreateRandomListMinMax(int min, int max, int cols, int rows, double *data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState_t state;
    hiprand_init(clock64(), tid, 0, &state);
    for (int i = tid; i < rows * cols; i += stride)
    {
        auto value = generateRandomNumber(min, max, &state);
        data[i] = value;
    }
}
// Kernel to multiply each element of data with a double value
__global__ void MultiplyData(double multiplier, int cols, int rows, double *data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < rows * cols; i += stride)
    {
        data[i] *= multiplier;
    }
}
__global__ void SetData(double val, int cols, int rows, double *data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < rows * cols; i += stride)
    {
        data[i] = val;
    }
}
