#include "hip/hip_runtime.h"
#include "tensor2d_kernel.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void RandomInRangeKernel(int min, int max, int cols, int rows, float **results)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        hiprandState state;
        hiprand_init(0, row * cols + col, 0, &state); // Initialize hiprand state

        // Generate a random value within the specified range
        float random_value = min + (max - min) * hiprand_uniform(&state);
        results[row][col] = random_value;
    }
}
